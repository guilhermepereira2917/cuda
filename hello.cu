
#include <hip/hip_runtime.h>
#include <stdio.h>

const int ARRAY_SIZE = 1000000;
__managed__ int vector_a[ARRAY_SIZE], vector_b[ARRAY_SIZE], vector_c[ARRAY_SIZE];

__global__ void add_threaded(int* a, int* b, int* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

void add_unthreaded(int* a, int* b, int* c) {
    for (int i = 0; i < ARRAY_SIZE; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    for (int i = 0; i < ARRAY_SIZE; i++) {
        vector_a[i] = i;
        vector_b[i] = ARRAY_SIZE - i;
    }

    hipEvent_t start, stop;

    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    add_threaded<<<1, ARRAY_SIZE>>>(vector_a, vector_b, vector_c);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time to generate threaded:    %3.6f ms \n", milliseconds);

    milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    add_unthreaded(vector_a, vector_b, vector_c);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time to generate unthreaded:  %3.6f ms \n", milliseconds);
}